#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// Loss computation-related kernels

#include <torch/extension.h>
#include <cstdint>
#include <cstdio>
#include "cuda_util.cuh"
#include "cubemap_util.cuh"

namespace {

const int TV_GRAD_CUDA_THREADS = 256;
const int MIN_BLOCKS_PER_SM = 4;

namespace device {

__device__ __inline__
void calculate_ray_scale(float ndc_coeffx,
                         float ndc_coeffy,
                         float z,
                         // float maxx,
                         // float maxy,
                         float maxz,
                         float* __restrict__ scale) {
    if (ndc_coeffx > 0.f) {
        // FF NDC

        // Normalized to [-1, 1] (with 0.5 padding)
        // const float x_norm = (x + 0.5) / maxx * 2 - 1;
        // const float y_norm = (y + 0.5) / maxy * 2 - 1;
        const float z_norm = (z + 0.5) / maxz * 2 - 1;

        // NDC distances
        const float disparity = (1 - z_norm) / 2.f; // in [0, 1]
        scale[0] = (ndc_coeffx * disparity);//maxx * 0.5f;
        scale[1] = (ndc_coeffy * disparity);//maxy * 0.5f;
        scale[2] = -((z_norm - 1.f + 2.f / maxz) * disparity) / (maxz * 0.5f);
    } else {
        scale[0] = 1.f; //maxx * 0.5f;
        scale[1] = 1.f; //maxy * 0.5f;
        scale[2] = 1.f; //maxz * 0.5f;
    }
}

// __device__ __inline__
// void approx_msi_scale(float msi_nlayers,
//         float msi_layer_id) {
//     // MSI (approximate)
//     const float radius = msi_nlayers /
//         (msi_nlayers - msi_layer_id - 0.5f);
//     const float full_thickness = msi_nlayers /
//         (msi_nlayers - msi_layer_id - 1.5f) - radius;
//     scale[0] = 1.f / full_thickness;
//     scale[1] = scale[0];
//     scale[2] = scale[0];
// }


#define CALCULATE_RAY_SCALE(out_name) float out_name[3]; \
    calculate_ray_scale( \
            ndc_coeffx, ndc_coeffy, \
            z, \
            links.size(2), \
            out_name)

__global__ void tv_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        float* __restrict__ out) {
    CUDA_GET_THREAD_ID_U64(tid, Q);

    typedef hipcub::BlockReduce<float, 1024> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    CALCULATE_RAY_SCALE(scaling);

    const float val000 = (links[x][y][z] >= 0 ?
                          data[links[x][y][z]][idx] : 0.f);
    const float null_val = (ignore_edge ? val000 : 0.f);
    const float val100 = (links[x + 1][y][z] >= 0 ?
                          data[links[x + 1][y][z]][idx] : null_val);
    const float val010 = (links[x][y + 1][z] >= 0 ?
                          data[links[x][y + 1][z]][idx] : null_val);
    const float val001 = (links[x][y][z + 1] >= 0 ?
                          data[links[x][y][z + 1]][idx] : null_val);
    const float dx = (val100 - val000) * scaling[0];
    const float dy = (val010 - val000) * scaling[1];
    const float dz = (val001 - val000) * scaling[2];
    const float tresult = sqrtf(1e-5f + dx * dx + dy * dy + dz * dz);

    const float bresult = BlockReduce(temp_storage).Sum(tresult);
    if (threadIdx.x == 0) {
        atomicAdd(out, bresult * scale);
    }
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    float dummy;
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    CALCULATE_RAY_SCALE(scaling);

    const float* dptr = data.data();
    const size_t ddim = data.size(1);
    float v000 = 0.f, v100 = 0.f, v010 = 0.f, v001 = 0.f;
    float* gptr000 = &dummy,
         * gptr100 = &dummy,
         * gptr010 = &dummy,
         * gptr001 = &dummy;

    if (links[x][y][z] >= 0) {
        const size_t lnk = links[x][y][z] * ddim + idx;
        v000 = dptr[lnk];
        gptr000 = grad_data + lnk;
    }
    if (links[x + 1][y][z] >= 0) {
        const size_t lnk = links[x + 1][y][z] * ddim + idx;
        v100 = dptr[lnk];
        gptr100 = grad_data + lnk;
    } else if (ignore_edge) v100 = v000;
    if (links[x][y + 1][z] >= 0) {
        const size_t lnk = links[x][y + 1][z] * ddim + idx;
        v010 = dptr[lnk];
        gptr010 = grad_data + lnk;
    } else if (ignore_edge) v010 = v000;
    if (links[x][y][z + 1] >= 0) {
        const size_t lnk = links[x][y][z + 1] * ddim + idx;
        v001 = dptr[lnk];
        gptr001 = grad_data + lnk;
    } else if (ignore_edge) v001 = v000;

    float dx = (v100 - v000);
    float dy = (v010 - v000);
    float dz = (v001 - v000);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);
    dx *= scaling[0];
    dy *= scaling[1];
    dz *= scaling[2];
    if (dx != 0.f) atomicAdd(gptr100, dx * idelta);
    if (dy != 0.f) atomicAdd(gptr010, dy * idelta);
    if (dz != 0.f) atomicAdd(gptr001, dz * idelta);
    atomicAdd(gptr000, -(dx + dy + dz) * idelta);
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_sparse_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        const int32_t* __restrict__ rand_cells,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        bool* __restrict__ mask_out,
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = rand_cells[tid / (end_dim - start_dim)];
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    const int32_t* __restrict__ links_ptr = &links[x][y][z];

    if (ignore_edge && *links_ptr == 0) return;

    CALCULATE_RAY_SCALE(scaling);

    const int offx = links.stride(0), offy = links.stride(1);

    const float v000 = links_ptr[0] >= 0 ? data[links_ptr[0]][idx] : 0.f;
    const float null_val = (ignore_edge ? v000 : 0.f);
    const float v001 = links_ptr[1] >= 0 ? data[links_ptr[1]][idx] : null_val,
                v010 = links_ptr[offy] >= 0 ? data[links_ptr[offy]][idx] : null_val,
                v100 = links_ptr[offx] >= 0 ? data[links_ptr[offx]][idx] : null_val;

    float dx = (v100 - v000);
    float dy = (v010 - v000);
    float dz = (v001 - v000);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);

    dx *= scaling[0];
    dy *= scaling[1];
    dz *= scaling[2];

#define MAYBE_ADD_SET(gp, val) if (links_ptr[gp] >= 0 && val != 0.f) { \
    atomicAdd(&grad_data[links_ptr[gp] * data.size(1) + idx], val * idelta); \
    if (mask_out != nullptr) { \
        mask_out[links_ptr[gp]] = true; \
    } \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(0, sm);
    MAYBE_ADD_SET(1, dz);
    MAYBE_ADD_SET(offy, dy);
    MAYBE_ADD_SET(offx, dx);

#undef MAYBE_ADD_SET
}

// Cauchy
// #define _LOGALPHA(x)  logf(1.0 + delta * x * x + 1e-3)
// #define _D_LOGALPHA(x)  (delta * 2 * x) / (1.0 + delta * x * x + 1e-3)

// Log alpha (NV)
#define _LOGALPHA(x)  logf(1.0 - expf(- delta * x) + 1e-3)
#define _D_LOGALPHA(x) ((delta * expf(-delta * fmaxf(x, 0)) * (x > 0.f)) / \
                         (1.0 - expf(-delta * fmaxf(x, 0)) + 1e-3))

__global__ void tv_logalpha_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        float* __restrict__ out) {
    CUDA_GET_THREAD_ID_U64(tid, Q);

    typedef hipcub::BlockReduce<float, 1024> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float val000 = (links[x][y][z] >= 0 ?
                          _LOGALPHA(data[links[x][y][z]][idx]) : 0.f);
    const float null_val = (ignore_edge ? val000 : 0.f);
    const float val100 = (links[x + 1][y][z] >= 0 ?
                          _LOGALPHA(data[links[x + 1][y][z]][idx]) : null_val);
    const float val010 = (links[x][y + 1][z] >= 0 ?
                          _LOGALPHA(data[links[x][y + 1][z]][idx]) : null_val);
    const float val001 = (links[x][y][z + 1] >= 0 ?
                          _LOGALPHA(data[links[x][y][z + 1]][idx]) : null_val);
    const float dx = val100 - val000;
    const float dy = val010 - val000;
    const float dz = val001 - val000;
    const float tresult = sqrtf(1e-5f + dx * dx + dy * dy + dz * dz);

    const float bresult = BlockReduce(temp_storage).Sum(tresult);
    if (threadIdx.x == 0) {
        atomicAdd(out, bresult * scale);
    }
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_logalpha_grad_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    float dummy;
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float* dptr = data.data();
    const size_t ddim = data.size(1);
    float v000 = 0.f, v100 = 0.f, v010 = 0.f, v001 = 0.f;
    float* gptr000 = &dummy,
         * gptr100 = &dummy,
         * gptr010 = &dummy,
         * gptr001 = &dummy;

    if (links[x][y][z] >= 0) {
        const size_t lnk = links[x][y][z] * ddim + idx;
        v000 = dptr[lnk];
        gptr000 = grad_data + lnk;
    }
    if (links[x + 1][y][z] >= 0) {
        const size_t lnk = links[x + 1][y][z] * ddim + idx;
        v100 = dptr[lnk];
        gptr100 = grad_data + lnk;
    } else if (ignore_edge) v100 = v000;
    if (links[x][y + 1][z] >= 0) {
        const size_t lnk = links[x][y + 1][z] * ddim + idx;
        v010 = dptr[lnk];
        gptr010 = grad_data + lnk;
    } else if (ignore_edge) v010 = v000;
    if (links[x][y][z + 1] >= 0) {
        const size_t lnk = links[x][y][z + 1] * ddim + idx;
        v001 = dptr[lnk];
        gptr001 = grad_data + lnk;
    } else if (ignore_edge) v001 = v000;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
    if (dx != 0.f) atomicAdd(gptr100, dx * idelta * _D_LOGALPHA(v100));
    if (dy != 0.f) atomicAdd(gptr010, dy * idelta * _D_LOGALPHA(v010));
    if (dz != 0.f) atomicAdd(gptr001, dz * idelta * _D_LOGALPHA(v001));
    atomicAdd(gptr000, -(dx + dy + dz) * idelta);
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_logalpha_grad_sparse_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        const int32_t* __restrict__ rand_cells,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        bool* __restrict__ mask_out,
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = rand_cells[tid / (end_dim - start_dim)];
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    const int32_t* __restrict__ links_ptr = &links[x][y][z];

    if (ignore_edge && *links_ptr == 0) return;
    const int offx = links.stride(0), offy = links.stride(1);

    const float v000 = links_ptr[0] >= 0 ? data[links_ptr[0]][idx] : 0.f;
    const float null_val = (ignore_edge ? v000 : 0.f);
    const float v001 = links_ptr[1] >= 0 ? data[links_ptr[1]][idx] : null_val,
                v010 = links_ptr[offy] >= 0 ? data[links_ptr[offy]][idx] : null_val,
                v100 = links_ptr[offx] >= 0 ? data[links_ptr[offx]][idx] : null_val;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
#define MAYBE_ADD_SET(gp, expr) { \
    float val = (expr);\
    if (links_ptr[gp] >= 0 && val != 0.f) { \
    atomicAdd(&grad_data[links_ptr[gp] * data.size(1) + idx], val * idelta); \
    if (mask_out != nullptr) { \
        mask_out[links_ptr[gp]] = true; \
    } \
} \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(0, sm * _D_LOGALPHA(v000));
    MAYBE_ADD_SET(1, dz * _D_LOGALPHA(v001));
    MAYBE_ADD_SET(offy, dy * _D_LOGALPHA(v010));
    MAYBE_ADD_SET(offx, dx * _D_LOGALPHA(v100));

#undef MAYBE_ADD_SET
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void msi_tv_grad_sparse_kernel(
        // (n_layers, 6, reso, reso, n_channels)
        const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> cubemap,
        const int32_t* __restrict__ rand_cells,
        float scale,
        float scale_last,
        size_t Q,
        // Output
        torch::PackedTensorAccessor32<bool, 4, torch::RestrictPtrTraits> cubemap_mask,
        torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> grad_cubemap) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int channel_id = tid % cubemap.size(4);
    const int msi_idx = rand_cells[tid / cubemap.size(4)];

    const int v = msi_idx % (cubemap.size(3) - 1);
    int tmp = msi_idx / (cubemap.size(3) - 1);
    const int u = tmp % (cubemap.size(2) - 1);
    tmp /= (cubemap.size(2) - 1);
    const int face_id = tmp % cubemap.size(1);
    const int layer_id = tmp / cubemap.size(1);

    const float v00 = cubemap[layer_id][face_id][u][v][channel_id];
    const float v01 = cubemap[layer_id][face_id][u][v + 1][channel_id];
    const float v10 = cubemap[layer_id][face_id][u + 1][v][channel_id];
    const float v_nxl = cubemap[layer_id + 1][face_id][u][v][channel_id];

    if (channel_id == cubemap.size(4) - 1) {
        scale = scale_last;
    }

    float dx = (v10 - v00);
    float dy = (v01 - v00);
    float dz = (v_nxl - v00);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);

    const float msi_nlayers = cubemap.size(0);

    const float radius = msi_nlayers / (msi_nlayers - layer_id - 0.5f);
    const float nxl_radius = msi_nlayers / (msi_nlayers - layer_id - 1.5f);
    float coord00[3], coord01[3], coord10[3], coord_nxl[3];
    invert_cubemap(u, v, radius, cubemap.size(2), coord00);
    invert_cubemap(u + 1, v, radius, cubemap.size(2), coord01);
    invert_cubemap(u, v + 1, radius, cubemap.size(2), coord10);
    invert_cubemap(u, v, nxl_radius + 1.f, cubemap.size(2), coord_nxl);

    xsuby3d(coord01, coord00);
    xsuby3d(coord10, coord00);
    xsuby3d(coord_nxl, coord00);
    dx *= _rnorm(coord01);
    dy *= _rnorm(coord10);
    dz *= _rnorm(coord_nxl);

#define MAYBE_ADD_SET(layer_id, face_id, u, v, val) if (val != 0.f) { \
    atomicAdd(&grad_cubemap[layer_id][face_id][u][v][channel_id], val * idelta); \
    if (cubemap_mask.size(0) > 0) \
        cubemap_mask[layer_id][face_id][u][v] = true; \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(layer_id, face_id, u, v, sm);
    MAYBE_ADD_SET(layer_id + 1, face_id, u, v, dz);
    MAYBE_ADD_SET(layer_id, face_id, u, v + 1, dy);
    MAYBE_ADD_SET(layer_id, face_id, u + 1, v, dx);
#undef MAYBE_ADD_SET
}

}  // namespace device
}  // namespace


torch::Tensor tv(torch::Tensor links, torch::Tensor data,
                 int start_dim, int end_dim,
                 bool use_logalpha,
                 float logalpha_delta,
                 bool ignore_edge,
                 float ndc_coeffx,
                 float ndc_coeffy) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int cuda_n_threads = 1024;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor result = torch::zeros({}, data.options());
    if (use_logalpha) {
        // TODO this should also use scaling
        device::tv_logalpha_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                1.f / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                result.data_ptr<float>());
    } else {
        device::tv_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                1.f / nl,
                Q,
                ignore_edge,
                ndc_coeffx, ndc_coeffy,
                // Output
                result.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
    return result;
}

void tv_grad(torch::Tensor links,
             torch::Tensor data,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             float ndc_coeffx,
             float ndc_coeffy,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    if (use_logalpha) {
        device::tv_logalpha_grad_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                grad_data.data_ptr<float>());
    } else {
        device::tv_grad_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                ignore_edge,
                ndc_coeffx, ndc_coeffy,
                // Output
                grad_data.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
}

void tv_grad_sparse(torch::Tensor links,
             torch::Tensor data,
             torch::Tensor rand_cells,
             torch::Tensor mask_out,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             float ndc_coeffx,
             float ndc_coeffy,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(mask_out);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    if (use_logalpha) {
        device::tv_logalpha_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                rand_cells.data_ptr<int32_t>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                (mask_out.dim() > 0) ? mask_out.data_ptr<bool>() : nullptr,
                grad_data.data_ptr<float>());
    } else {
        device::tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                rand_cells.data_ptr<int32_t>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                ignore_edge,
                ndc_coeffx, ndc_coeffy,
                // Output
                (mask_out.dim() > 0) ? mask_out.data_ptr<bool>() : nullptr,
                grad_data.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
}

void msi_tv_grad_sparse(torch::Tensor cubemap,
             torch::Tensor rand_cells,
             torch::Tensor mask_out,
             float scale,
             float scale_last,
             torch::Tensor grad_cubemap) {
    DEVICE_GUARD(cubemap);
    CHECK_INPUT(cubemap);
    CHECK_INPUT(grad_cubemap);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(mask_out);
    TORCH_CHECK(cubemap.is_floating_point());
    TORCH_CHECK(grad_cubemap.is_floating_point());
    TORCH_CHECK(cubemap.ndimension() == 5);
    TORCH_CHECK(grad_cubemap.ndimension() == 5);
    TORCH_CHECK(mask_out.ndimension() == 4);

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * cubemap.size(4);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::msi_tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
            cubemap.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
            rand_cells.data_ptr<int32_t>(),
            scale / nl,
            scale_last / nl,
            Q,
            // Output
            mask_out.packed_accessor32<bool, 4, torch::RestrictPtrTraits>(),
            grad_cubemap.packed_accessor32<float, 5, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
}
