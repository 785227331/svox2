#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// Loss computation-related kernels

#include <torch/extension.h>
#include <cstdint>
#include <cstdio>
#include "cuda_util.cuh"

namespace {

const int TV_GRAD_CUDA_THREADS = 256;
const int MIN_BLOCKS_PER_SM = 4;

namespace device {
__global__ void tv_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        // Output
        float* __restrict__ out) {
    CUDA_GET_THREAD_ID_U64(tid, Q);

    typedef hipcub::BlockReduce<float, 1024> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float val000 = (links[x][y][z] >= 0 ?
                          data[links[x][y][z]][idx] : 0.f);
    const float null_val = (ignore_edge ? val000 : 0.f);
    const float val100 = (links[x + 1][y][z] >= 0 ?
                          data[links[x + 1][y][z]][idx] : null_val);
    const float val010 = (links[x][y + 1][z] >= 0 ?
                          data[links[x][y + 1][z]][idx] : null_val);
    const float val001 = (links[x][y][z + 1] >= 0 ?
                          data[links[x][y][z + 1]][idx] : null_val);
    const float dx = val100 - val000;
    const float dy = val010 - val000;
    const float dz = val001 - val000;
    const float tresult = sqrtf(1e-5f + dx * dx + dy * dy + dz * dz);

    const float bresult = BlockReduce(temp_storage).Sum(tresult);
    if (threadIdx.x == 0) {
        atomicAdd(out, bresult * scale);
    }
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        // Output
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    float dummy;
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float* dptr = data.data();
    const size_t ddim = data.size(1);
    float v000 = 0.f, v100 = 0.f, v010 = 0.f, v001 = 0.f;
    float* gptr000 = &dummy,
         * gptr100 = &dummy,
         * gptr010 = &dummy,
         * gptr001 = &dummy;

    if (links[x][y][z] >= 0) {
        const size_t lnk = links[x][y][z] * ddim + idx;
        v000 = dptr[lnk];
        gptr000 = grad_data + lnk;
    }
    if (links[x + 1][y][z] >= 0) {
        const size_t lnk = links[x + 1][y][z] * ddim + idx;
        v100 = dptr[lnk];
        gptr100 = grad_data + lnk;
    } else if (ignore_edge) v100 = v000;
    if (links[x][y + 1][z] >= 0) {
        const size_t lnk = links[x][y + 1][z] * ddim + idx;
        v010 = dptr[lnk];
        gptr010 = grad_data + lnk;
    } else if (ignore_edge) v010 = v000;
    if (links[x][y][z + 1] >= 0) {
        const size_t lnk = links[x][y][z + 1] * ddim + idx;
        v001 = dptr[lnk];
        gptr001 = grad_data + lnk;
    } else if (ignore_edge) v001 = v000;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
    if (dx != 0.f) atomicAdd(gptr100, dx * idelta);
    if (dy != 0.f) atomicAdd(gptr010, dy * idelta);
    if (dz != 0.f) atomicAdd(gptr001, dz * idelta);
    atomicAdd(gptr000, -(dx + dy + dz) * idelta);
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_sparse_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        const int32_t* __restrict__ rand_cells,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        // Output
        bool* __restrict__ mask_out,
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = rand_cells[tid / (end_dim - start_dim)];
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    const int32_t* __restrict__ links_ptr = &links[x][y][z];

    if (ignore_edge && *links_ptr == 0) return;
    const int offx = links.stride(0), offy = links.stride(1);

    const float v000 = links_ptr[0] >= 0 ? data[links_ptr[0]][idx] : 0.f;
    const float null_val = (ignore_edge ? v000 : 0.f);
    const float v001 = links_ptr[1] >= 0 ? data[links_ptr[1]][idx] : null_val,
                v010 = links_ptr[offy] >= 0 ? data[links_ptr[offy]][idx] : null_val,
                v100 = links_ptr[offx] >= 0 ? data[links_ptr[offx]][idx] : null_val;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
#define MAYBE_ADD_SET(gp, val) if (links_ptr[gp] >= 0 && val != 0.f) { \
    atomicAdd(&grad_data[links_ptr[gp] * data.size(1) + idx], val * idelta); \
    if (mask_out != nullptr) { \
        mask_out[links_ptr[gp]] = true; \
    } \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(0, sm);
    MAYBE_ADD_SET(1, dz);
    MAYBE_ADD_SET(offy, dy);
    MAYBE_ADD_SET(offx, dx);

#undef MAYBE_ADD_SET
}

// Cauchy
// #define _LOGALPHA(x)  logf(1.0 + delta * x * x + 1e-3)
// #define _D_LOGALPHA(x)  (delta * 2 * x) / (1.0 + delta * x * x + 1e-3)

// Log alpha (NV)
#define _LOGALPHA(x)  logf(1.0 - expf(- delta * x) + 1e-3)
#define _D_LOGALPHA(x) ((delta * expf(-delta * fmaxf(x, 0)) * (x > 0.f)) / \
                         (1.0 - expf(-delta * fmaxf(x, 0)) + 1e-3))

__global__ void tv_logalpha_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        float* __restrict__ out) {
    CUDA_GET_THREAD_ID_U64(tid, Q);

    typedef hipcub::BlockReduce<float, 1024> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float val000 = (links[x][y][z] >= 0 ?
                          _LOGALPHA(data[links[x][y][z]][idx]) : 0.f);
    const float null_val = (ignore_edge ? val000 : 0.f);
    const float val100 = (links[x + 1][y][z] >= 0 ?
                          _LOGALPHA(data[links[x + 1][y][z]][idx]) : null_val);
    const float val010 = (links[x][y + 1][z] >= 0 ?
                          _LOGALPHA(data[links[x][y + 1][z]][idx]) : null_val);
    const float val001 = (links[x][y][z + 1] >= 0 ?
                          _LOGALPHA(data[links[x][y][z + 1]][idx]) : null_val);
    const float dx = val100 - val000;
    const float dy = val010 - val000;
    const float dz = val001 - val000;
    const float tresult = sqrtf(1e-5f + dx * dx + dy * dy + dz * dz);

    const float bresult = BlockReduce(temp_storage).Sum(tresult);
    if (threadIdx.x == 0) {
        atomicAdd(out, bresult * scale);
    }
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_logalpha_grad_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    float dummy;
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    const float* dptr = data.data();
    const size_t ddim = data.size(1);
    float v000 = 0.f, v100 = 0.f, v010 = 0.f, v001 = 0.f;
    float* gptr000 = &dummy,
         * gptr100 = &dummy,
         * gptr010 = &dummy,
         * gptr001 = &dummy;

    if (links[x][y][z] >= 0) {
        const size_t lnk = links[x][y][z] * ddim + idx;
        v000 = dptr[lnk];
        gptr000 = grad_data + lnk;
    }
    if (links[x + 1][y][z] >= 0) {
        const size_t lnk = links[x + 1][y][z] * ddim + idx;
        v100 = dptr[lnk];
        gptr100 = grad_data + lnk;
    } else if (ignore_edge) v100 = v000;
    if (links[x][y + 1][z] >= 0) {
        const size_t lnk = links[x][y + 1][z] * ddim + idx;
        v010 = dptr[lnk];
        gptr010 = grad_data + lnk;
    } else if (ignore_edge) v010 = v000;
    if (links[x][y][z + 1] >= 0) {
        const size_t lnk = links[x][y][z + 1] * ddim + idx;
        v001 = dptr[lnk];
        gptr001 = grad_data + lnk;
    } else if (ignore_edge) v001 = v000;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
    if (dx != 0.f) atomicAdd(gptr100, dx * idelta * _D_LOGALPHA(v100));
    if (dy != 0.f) atomicAdd(gptr010, dy * idelta * _D_LOGALPHA(v010));
    if (dz != 0.f) atomicAdd(gptr001, dz * idelta * _D_LOGALPHA(v001));
    atomicAdd(gptr000, -(dx + dy + dz) * idelta);
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_logalpha_grad_sparse_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        const int32_t* __restrict__ rand_cells,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        float delta,
        bool ignore_edge,
        // Output
        bool* __restrict__ mask_out,
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = rand_cells[tid / (end_dim - start_dim)];
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    const int32_t* __restrict__ links_ptr = &links[x][y][z];

    if (ignore_edge && *links_ptr == 0) return;
    const int offx = links.stride(0), offy = links.stride(1);

    const float v000 = links_ptr[0] >= 0 ? data[links_ptr[0]][idx] : 0.f;
    const float null_val = (ignore_edge ? v000 : 0.f);
    const float v001 = links_ptr[1] >= 0 ? data[links_ptr[1]][idx] : null_val,
                v010 = links_ptr[offy] >= 0 ? data[links_ptr[offy]][idx] : null_val,
                v100 = links_ptr[offx] >= 0 ? data[links_ptr[offx]][idx] : null_val;

    const float dx = v100 - v000;
    const float dy = v010 - v000;
    const float dz = v001 - v000;
    const float idelta = scale * rsqrtf(1e-5f + dx * dx + dy * dy + dz * dz);
#define MAYBE_ADD_SET(gp, expr) { \
    float val = (expr);\
    if (links_ptr[gp] >= 0 && val != 0.f) { \
    atomicAdd(&grad_data[links_ptr[gp] * data.size(1) + idx], val * idelta); \
    if (mask_out != nullptr) { \
        mask_out[links_ptr[gp]] = true; \
    } \
} \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(0, sm * _D_LOGALPHA(v000));
    MAYBE_ADD_SET(1, dz * _D_LOGALPHA(v001));
    MAYBE_ADD_SET(offy, dy * _D_LOGALPHA(v010));
    MAYBE_ADD_SET(offx, dx * _D_LOGALPHA(v100));

#undef MAYBE_ADD_SET
}

}  // namespace device
}  // namespace


torch::Tensor tv(torch::Tensor links, torch::Tensor data,
                 int start_dim, int end_dim,
                 bool use_logalpha,
                 float logalpha_delta,
                 bool ignore_edge) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int cuda_n_threads = 1024;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor result = torch::zeros({}, data.options());
    if (use_logalpha) {
        device::tv_logalpha_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                1.f / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                result.data_ptr<float>());
    } else {
        device::tv_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                1.f / nl,
                Q,
                ignore_edge,
                // Output
                result.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
    return result;
}

void tv_grad(torch::Tensor links,
             torch::Tensor data,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    if (use_logalpha) {
        device::tv_logalpha_grad_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                grad_data.data_ptr<float>());
    } else {
        device::tv_grad_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                ignore_edge,
                // Output
                grad_data.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
}

void tv_grad_sparse(torch::Tensor links,
             torch::Tensor data,
             torch::Tensor rand_cells,
             torch::Tensor mask_out,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(mask_out);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    if (use_logalpha) {
        device::tv_logalpha_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                rand_cells.data_ptr<int32_t>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                logalpha_delta,
                ignore_edge,
                // Output
                (mask_out.dim() > 0) ? mask_out.data_ptr<bool>() : nullptr,
                grad_data.data_ptr<float>());
    } else {
        device::tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
                links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
                data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                rand_cells.data_ptr<int32_t>(),
                start_dim,
                end_dim,
                scale / nl,
                Q,
                ignore_edge,
                // Output
                (mask_out.dim() > 0) ? mask_out.data_ptr<bool>() : nullptr,
                grad_data.data_ptr<float>());
    }
    CUDA_CHECK_ERRORS;
}
