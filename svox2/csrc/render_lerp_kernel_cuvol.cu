#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
#include <torch/extension.h>
#include "cuda_util.cuh"
#include "data_spec_packed.cuh"
#include "render_util.cuh"
#include "cubemap_util.cuh"

#include <iostream>
#include <cstdint>
#include <tuple>

namespace {
const int WARP_SIZE = 32;

const int TRACE_RAY_CUDA_THREADS = 128;
const int TRACE_RAY_CUDA_RAYS_PER_BLOCK = TRACE_RAY_CUDA_THREADS / WARP_SIZE;

const int TRACE_RAY_BKWD_CUDA_THREADS = 128;
const int TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK = TRACE_RAY_BKWD_CUDA_THREADS / WARP_SIZE;

const int MIN_BLOCKS_PER_SM = 8;

const int TRACE_RAY_BG_CUDA_THREADS = 256;
const int MIN_BG_BLOCKS_PER_SM = 4;
typedef hipcub::WarpReduce<float> WarpReducef;

namespace device {


// * For ray rendering
__device__ __inline__ void trace_ray_cuvol(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        float* __restrict__ sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        float* __restrict__ out,
        float* __restrict__ out_log_transmit) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;

    if (ray.tmin > ray.tmax) {
        out[lane_colorgrp] = (grid.background_nlayers == 0) ? opt.background_brightness : 0.f;
        if (out_log_transmit != nullptr) {
            *out_log_transmit = 0.f;
        }
        return;
    }

    float t = ray.tmin;
    float outv = 0.f;

    float log_transmit = 0.f;
    // printf("tmin %f, tmax %f \n", ray.tmin, ray.tmax);

    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }
        // if (lane_id == 0) {
        //     printf("[%d %d %d] [%f %f %f] %f\n",
        //             ray.l[0], ray.l[1], ray.l[2],
        //             ray.pos[0], ray.pos[1], ray.pos[2], log_transmit);
        // }

        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);

        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }
        float sigma = trilerp_cuvol_one(
                grid.links, grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);
        if (opt.last_sample_opaque && t + opt.step_size > ray.tmax) {
            sigma += 1e9;
        }
        if (opt.randomize && opt.random_sigma_std > 0.0) sigma += ray.rng.randn() * opt.random_sigma_std;
        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            lane_color *= sphfunc_val[lane_colorgrp_id];

            const float pcnt = ray.world_step * sigma;
            const float weight = _EXP(log_transmit) * (1.f - _EXP(-pcnt));
            log_transmit -= pcnt;

            float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           lane_color, lane_colorgrp_id == 0);
            outv += weight * fmaxf(lane_color_total + 0.5f, 0.f);  // Clamp to [+0, infty)
            if (_EXP(log_transmit) < opt.stop_thresh) {
                log_transmit = -1e3f;
                break;
            }
        }
        t += opt.step_size;
    }

    if (grid.background_nlayers == 0) {
        outv += _EXP(log_transmit) * opt.background_brightness;
    }
    if (lane_colorgrp_id == 0) {
        if (out_log_transmit != nullptr) {
            *out_log_transmit = log_transmit;
        }
        out[lane_colorgrp] = outv;
    }
}

__device__ __inline__ void trace_ray_cuvol_backward(
        const PackedSparseGridSpec& __restrict__ grid,
        const float* __restrict__ grad_output,
        const float* __restrict__ color_cache,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        const float* __restrict__ sphfunc_val,
        float* __restrict__ grad_sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        float log_transmit_in,
        float beta_loss,
        float sparsity_loss,
        PackedGridOutputGrads& __restrict__ grads,
        float* __restrict__ accum_out,
        float* __restrict__ log_transmit_out
        ) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;
    const uint32_t leader_mask = 1U | (1U << grid.basis_dim) | (1U << (2 * grid.basis_dim));

    float accum = fmaf(color_cache[0], grad_output[0],
                      fmaf(color_cache[1], grad_output[1],
                           color_cache[2] * grad_output[2]));

    if (beta_loss > 0.f) {
        const float transmit_in = _EXP(log_transmit_in);
        beta_loss *= (1 - transmit_in / (1 - transmit_in + 1e-3)); // d beta_loss / d log_transmit_in
        accum += beta_loss;
        // Interesting how this loss turns out, kinda nice?
    }

    if (ray.tmin > ray.tmax) {
        if (accum_out != nullptr) { *accum_out = accum; }
        if (log_transmit_out != nullptr) { *log_transmit_out = 0.f; }
        // printf("accum_end_fg_fast=%f\n", accum);
        return;
    }
    float t = ray.tmin;

    const float gout = grad_output[lane_colorgrp];

    float log_transmit = 0.f;

    // remat samples
    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }
        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);
        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }

        float sigma = trilerp_cuvol_one(
                grid.links,
                grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);
        if (opt.last_sample_opaque && t + opt.step_size > ray.tmax) {
            sigma += 1e9;
        }
        if (opt.randomize && opt.random_sigma_std > 0.0) sigma += ray.rng.randn() * opt.random_sigma_std;
        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            float weighted_lane_color = lane_color * sphfunc_val[lane_colorgrp_id];

            const float pcnt = ray.world_step * sigma;
            const float weight = _EXP(log_transmit) * (1.f - _EXP(-pcnt));
            log_transmit -= pcnt;

            const float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           weighted_lane_color, lane_colorgrp_id == 0) + 0.5f;
            float total_color = fmaxf(lane_color_total, 0.f);
            float color_in_01 = total_color == lane_color_total;
            total_color *= gout; // Clamp to [+0, infty)

            float total_color_c1 = __shfl_sync(leader_mask, total_color, grid.basis_dim);
            total_color += __shfl_sync(leader_mask, total_color, 2 * grid.basis_dim);
            total_color += total_color_c1;

            color_in_01 = __shfl_sync((1U << grid.sh_data_dim) - 1, color_in_01, lane_colorgrp * grid.basis_dim);
            const float grad_common = weight * color_in_01 * gout;
            const float curr_grad_color = sphfunc_val[lane_colorgrp_id] * grad_common;

            if (grid.basis_type != BASIS_TYPE_SH) {
                float curr_grad_sphfunc = lane_color * grad_common;
                const float curr_grad_up2 = __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, 2 * grid.basis_dim);
                curr_grad_sphfunc += __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, grid.basis_dim);
                curr_grad_sphfunc += curr_grad_up2;
                if (lane_id < grid.basis_dim) {
                    grad_sphfunc_val[lane_id] += curr_grad_sphfunc;
                }
            }

            accum -= weight * total_color;
            float curr_grad_sigma = ray.world_step * (
                    total_color * _EXP(log_transmit) - accum);
            if (sparsity_loss > 0.f) {
                // Cauchy version (from SNeRG)
                curr_grad_sigma += sparsity_loss * (4 * sigma / (1 + 2 * (sigma * sigma)));

                // Alphs version (from PlenOctrees)
                // curr_grad_sigma += sparsity_loss * _EXP(-pcnt) * ray.world_step;
            }
            trilerp_backward_cuvol_one(grid.links, grads.grad_sh_out,
                    grid.stride_x,
                    grid.size[2],
                    grid.sh_data_dim,
                    ray.l, ray.pos,
                    curr_grad_color, lane_id);
            if (lane_id == 0) {
                trilerp_backward_cuvol_one_density(
                        grid.links,
                        grads.grad_density_out,
                        grads.mask_out,
                        grid.stride_x,
                        grid.size[2],
                        ray.l, ray.pos, curr_grad_sigma);
            }
            if (_EXP(log_transmit) < opt.stop_thresh) {
                break;
            }
        }
        t += opt.step_size;
    }
    if (lane_id == 0) {
        if (accum_out != nullptr) {
            // Cancel beta loss out in case of background
            accum -= beta_loss;
            *accum_out = accum;
        }
        if (log_transmit_out != nullptr) { *log_transmit_out = log_transmit; }
        // printf("accum_end_fg=%f\n", accum);
        // printf("log_transmit_fg=%f\n", log_transmit);
    }
}


__device__ __inline__ void render_background_forward(
            const PackedSparseGridSpec& __restrict__ grid,
            SingleRaySpec& __restrict__ ray,
            const RenderOptions& __restrict__ opt,
            float log_transmit,
            float* __restrict__ out
        ) {

    ConcentricSpheresIntersector csi(ray.origin, ray.dir);

    const float inner_radius = fmaxf(_dist_ray_to_origin(ray.origin, ray.dir) + 1e-3f, 1.f);
    float t, t_last;
    const int n_steps = int(grid.background_nlayers / opt.step_size) + 2;

    csi.intersect(inner_radius, &t_last);

    // printf("RAY o[%f,%f,%f] d[%f,%f,%f] rad=%f, t_last=%f ws=%f\n",
    //         ray.origin[0], ray.origin[1], ray.origin[2],
    //         ray.dir[0], ray.dir[1], ray.dir[2],
    //         inner_radius,
    //         t_last, ray.world_step);

    float outv[3] = {0.f, 0.f, 0.f};
    const int cubemap_step = 6 * grid.background_reso * grid.background_reso * /*n_channels*/ 4;
    const float layer_scale = (float(grid.background_nlayers - 1) / (n_steps + 1));
    for (int i = 0; i < n_steps; ++i) {
        float r = n_steps / (n_steps - i - 0.5);
        if (r < inner_radius || !csi.intersect(r, &t)) continue;
        const float t_mid = (t + t_last) * 0.5f;

#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t_mid, ray.dir[j], ray.origin[j]);
        }
        const float normalized_inv_radius = fminf((i + 1.f) * layer_scale,
                                                  grid.background_nlayers - 1);
        int layerid = min((int)floorf(normalized_inv_radius), grid.background_nlayers - 2);
        const float interp_wt = normalized_inv_radius - layerid;
        const float* __restrict__ cubemap_data = grid.background_cubemap + cubemap_step * layerid;

        const CubemapCoord coord = dir_to_cubemap_coord(ray.pos,
                grid.background_reso,
                /* EAC */ true);
        const CubemapBilerpQuery query = cubemap_build_query(coord,
                grid.background_reso);

        float sigma = multi_cubemap_sample(
                cubemap_data,
                cubemap_data + cubemap_step,
                query,
                interp_wt,
                grid.background_reso,
                /*n_channels*/ 4,
                3);
        // printf("SAMP p[%f,%f,%f] invr=%f layerid=%d interp_wt=%f sigma=%f log_li=%f\n",
        //         ray.pos[0], ray.pos[1], ray.pos[2],
        //         normalized_inv_radius,
        //         layerid,
        //         interp_wt,
        //         sigma,
        //         log_transmit);
        if (opt.randomize && opt.random_sigma_std_background > 0.0)
            sigma += ray.rng.randn() * opt.random_sigma_std_background;
        if (sigma > opt.sigma_thresh) {
            const float pcnt = (t - t_last) * ray.world_step * sigma;
            const float weight = _EXP(log_transmit) * (1.f - _EXP(-pcnt));
            log_transmit -= pcnt;
#pragma unroll 3
            for (int i = 0; i < 3; ++i) {
                const float color = multi_cubemap_sample(
                        cubemap_data,
                        cubemap_data + cubemap_step,
                        query,
                        interp_wt,
                        grid.background_reso,
                        /*n_channels*/ 4,
                        i) * C0;
                // printf("%f %d: %f %f\n", r, i, sigma, fmaxf(color + 0.5f, 0.f));
                outv[i] += weight * fmaxf(color + 0.5f, 0.f);  // Clamp to [+0, infty)
            }
            if (_EXP(log_transmit) < opt.stop_thresh) {
                break;
            }
        }
        if (cubemap_data != nullptr)
            cubemap_data += cubemap_step;
        t_last = t;
    }
#pragma unroll 3
    for (int i = 0; i < 3; ++i) {
        out[i] += outv[i] + _EXP(log_transmit) * opt.background_brightness;
    }
}

__device__ __inline__ void render_background_backward(
            const PackedSparseGridSpec& __restrict__ grid,
            const float* __restrict__ grad_output,
            SingleRaySpec& __restrict__ ray,
            const RenderOptions& __restrict__ opt,
            float log_transmit,
            float accum,
            PackedGridOutputGrads& __restrict__ grads
        ) {
    // printf("accum_init=%f\n", accum);
    // printf("log_transmit_init=%f\n", log_transmit);
    ConcentricSpheresIntersector csi(ray.origin, ray.dir);

    float t, t_last;
    const int n_steps = int(grid.background_nlayers / opt.step_size) + 2;
    const int cubemap_step = 6 * grid.background_reso * grid.background_reso;

    const float inner_radius = fmaxf(_dist_ray_to_origin(ray.origin, ray.dir) + 1e-3f, 1.f);
    csi.intersect(inner_radius, &t_last);
    const float layer_scale = (float(grid.background_nlayers - 1) / (n_steps + 1));

    for (int i = 0; i < n_steps; ++i) {
        float r = n_steps / (n_steps - i - 0.5);

        if (r < inner_radius || !csi.intersect(r, &t)) continue;

        const float t_mid = (t + t_last) * 0.5f;
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t_mid, ray.dir[j], ray.origin[j]);
        }

        const float normalized_inv_radius = fminf((i + 1.f) * layer_scale,
                                                  grid.background_nlayers - 1);
        int layerid = min((int)floorf(normalized_inv_radius), grid.background_nlayers - 2);
        const float interp_wt = normalized_inv_radius - layerid;
        const float* __restrict__ cubemap_data = grid.background_cubemap + cubemap_step * 4 * layerid;
        float* __restrict__ grad_cubemap_data = grads.grad_background_out == nullptr ? nullptr :
                                   (grads.grad_background_out + cubemap_step * 4 * layerid);
        bool* __restrict__ mask_cubemap_ptr = grads.mask_background_out == nullptr ?  nullptr :
                                   (grads.mask_background_out + cubemap_step * layerid);

        const CubemapCoord coord = dir_to_cubemap_coord(ray.pos,
                grid.background_reso, /* EAC */ true);
        const CubemapBilerpQuery query = cubemap_build_query(coord,
                grid.background_reso);

        float sigma = multi_cubemap_sample(
                cubemap_data,
                cubemap_data + cubemap_step * 4,
                query,
                interp_wt,
                grid.background_reso,
                /*n_channels*/ 4,
                3);
        if (opt.randomize && opt.random_sigma_std_background > 0.0)
            sigma += ray.rng.randn() * opt.random_sigma_std_background;
        if (sigma > opt.sigma_thresh) {
            float total_color = 0.f;
            const float pcnt = ray.world_step * (t - t_last) * sigma;
            const float weight = _EXP(log_transmit) * (1.f - _EXP(-pcnt));
            log_transmit -= pcnt;

            for (int i = 0; i < 3; ++i) {
                const float color = multi_cubemap_sample(
                        cubemap_data,
                        cubemap_data + cubemap_step * 4,
                        query,
                        interp_wt,
                        grid.background_reso,
                        /*n_channels*/ 4,
                        i) * C0 + 0.5f;
                total_color += fmaxf(color, 0.f) * grad_output[i];
                if (color > 0.f) {
                    const float curr_grad_color = C0 * weight * grad_output[i];
                    multi_cubemap_sample_backward(
                            grad_cubemap_data,
                            grad_cubemap_data + cubemap_step * 4,
                            query,
                            interp_wt,
                            grid.background_reso,
                            4,
                            curr_grad_color,
                            i);
                }
            }

            accum -= weight * total_color;
            float curr_grad_sigma = ray.world_step * (t - t_last) * (
                    total_color * _EXP(log_transmit) - accum);

            multi_cubemap_sample_backward(
                    grad_cubemap_data,
                    grad_cubemap_data + cubemap_step * 4,
                    query,
                    interp_wt,
                    grid.background_reso,
                    4,
                    curr_grad_sigma,
                    3,
                    mask_cubemap_ptr,
                    mask_cubemap_ptr + cubemap_step);

            if (_EXP(log_transmit) < opt.stop_thresh) {
                break;
            }
        }
        t_last = t;
    }
}

// BEGIN KERNELS

__launch_bounds__(TRACE_RAY_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_kernel(
        PackedSparseGridSpec grid,
        PackedRaysSpec rays,
        RenderOptions opt,
        torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> out,
        float* __restrict__ log_transmit_out = nullptr) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
            rays.dirs[ray_id].data());
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 ray_spec[ray_blk_id].dir,
                 sphfunc_val[ray_blk_id]);
    ray_find_bounds(ray_spec[ray_blk_id], grid, opt, ray_id);
    __syncwarp((1U << grid.sh_data_dim) - 1);

    trace_ray_cuvol(
        grid,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        out[ray_id].data(),
        log_transmit_out == nullptr ? nullptr : log_transmit_out + ray_id);
}

__launch_bounds__(TRACE_RAY_BKWD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_backward_kernel(
    PackedSparseGridSpec grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
    PackedRaysSpec rays,
    RenderOptions opt,
    bool grad_out_is_rgb,
    const float* __restrict__ log_transmit_in,
    float beta_loss,
    float sparsity_loss,
    PackedGridOutputGrads grads,
    float* __restrict__ accum_out = nullptr,
    float* __restrict__ log_transmit_out = nullptr) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK][10];
    __shared__ float grad_sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
                             rays.dirs[ray_id].data());
    const float vdir[3] = {ray_spec[ray_blk_id].dir[0],
                     ray_spec[ray_blk_id].dir[1],
                     ray_spec[ray_blk_id].dir[2] };
    if (lane_id < grid.basis_dim) {
        grad_sphfunc_val[ray_blk_id][lane_id] = 0.f;
    }
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 vdir, sphfunc_val[ray_blk_id]);
    if (lane_id == 0) {
        ray_find_bounds(ray_spec[ray_blk_id], grid, opt, ray_id);
    }

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[ray_id * 3 + i] - grad_output[ray_id * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[ray_id * 3 + i];
        }
    }

    __syncwarp((1U << grid.sh_data_dim) - 1);
    trace_ray_cuvol_backward(
        grid,
        grad_out,
        color_cache + ray_id * 3,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        grad_sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        log_transmit_in == nullptr ? 0.f : log_transmit_in[ray_id],
        beta_loss,
        sparsity_loss,
        grads,
        accum_out == nullptr ? nullptr : accum_out + ray_id,
        log_transmit_out == nullptr ? nullptr : log_transmit_out + ray_id);
    calc_sphfunc_backward(
                 grid, lane_id,
                 ray_id,
                 vdir,
                 sphfunc_val[ray_blk_id],
                 grad_sphfunc_val[ray_blk_id],
                 grads.grad_basis_out);
}

__launch_bounds__(TRACE_RAY_BG_CUDA_THREADS, MIN_BG_BLOCKS_PER_SM)
__global__ void render_background_kernel(
        PackedSparseGridSpec grid,
        PackedRaysSpec rays,
        RenderOptions opt,
        const float* __restrict__ log_transmit,
        // Outputs
        torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> out) {
    CUDA_GET_THREAD_ID(ray_id, int(rays.origins.size(0)));
    if (log_transmit[ray_id] < -25.f) return;
    SingleRaySpec ray_spec(rays.origins[ray_id].data(), rays.dirs[ray_id].data());
    ray_find_bounds_bg(ray_spec, grid, opt, ray_id);
    render_background_forward(
        grid,
        ray_spec,
        opt,
        log_transmit[ray_id],
        out[ray_id].data());
}

__launch_bounds__(TRACE_RAY_BG_CUDA_THREADS, MIN_BG_BLOCKS_PER_SM)
__global__ void render_background_backward_kernel(
        PackedSparseGridSpec grid,
        const float* __restrict__ grad_output,
        const float* __restrict__ color_cache,
        PackedRaysSpec rays,
        RenderOptions opt,
        const float* __restrict__ log_transmit,
        const float* __restrict__ accum,
        bool grad_out_is_rgb,
        // Outputs
        PackedGridOutputGrads grads) {
    CUDA_GET_THREAD_ID(ray_id, int(rays.origins.size(0)));
    if (log_transmit[ray_id] < -25.f) return;
    SingleRaySpec ray_spec(rays.origins[ray_id].data(), rays.dirs[ray_id].data());
    ray_find_bounds_bg(ray_spec, grid, opt, ray_id);

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[ray_id * 3 + i] - grad_output[ray_id * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[ray_id * 3 + i];
        }
    }

    render_background_backward(
        grid,
        grad_out,
        ray_spec,
        opt,
        log_transmit[ray_id],
        accum[ray_id],
        grads);
}

}  // namespace device

torch::Tensor _get_empty_1d(const torch::Tensor& origins) {
    auto options =
        torch::TensorOptions()
        .dtype(origins.dtype())
        .layout(torch::kStrided)
        .device(origins.device())
        .requires_grad(false);
    return torch::empty({origins.size(0)}, options);
}

}  // namespace

torch::Tensor volume_render_cuvol(SparseGridSpec& grid, RaysSpec& rays, RenderOptions& opt) {
    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();


    const auto Q = rays.origins.size(0);

    torch::Tensor results = torch::empty_like(rays.origins);

    bool use_background = grid.background_cubemap.size(0) > 0;
    torch::Tensor log_transmit;
    if (use_background) {
        log_transmit = _get_empty_1d(rays.origins);
    }

    {
        const int cuda_n_threads = TRACE_RAY_CUDA_THREADS;
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads);
        device::render_ray_kernel<<<blocks, cuda_n_threads>>>(
                grid, rays, opt,
                // Output
                results.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
                use_background ? log_transmit.data_ptr<float>() : nullptr);
    }

    if (use_background) {
        // printf("RENDER BG\n");
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, TRACE_RAY_BG_CUDA_THREADS);
        device::render_background_kernel<<<blocks, TRACE_RAY_BG_CUDA_THREADS>>>(
                grid,
                rays,
                opt,
                log_transmit.data_ptr<float>(),
                results.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    }

    CUDA_CHECK_ERRORS;
    return results;
}

void volume_render_cuvol_backward(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor grad_out,
        torch::Tensor color_cache,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    bool use_background = grid.background_cubemap.size(0) > 0;
    torch::Tensor log_transmit, accum;
    if (use_background) {
        log_transmit = _get_empty_1d(rays.origins);
        accum = _get_empty_1d(rays.origins);
    }

    {
        const int cuda_n_threads_render_backward = TRACE_RAY_BKWD_CUDA_THREADS;
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads_render_backward);
        device::render_ray_backward_kernel<<<blocks,
            cuda_n_threads_render_backward>>>(
                    grid,
                    grad_out.data_ptr<float>(),
                    color_cache.data_ptr<float>(),
                    rays, opt,
                    false,
                    nullptr,
                    0.f,
                    0.f,
                    // Output
                    grads,
                    use_background ? accum.data_ptr<float>() : nullptr,
                    use_background ? log_transmit.data_ptr<float>() : nullptr);
    }

    if (use_background) {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, TRACE_RAY_BG_CUDA_THREADS);
        device::render_background_backward_kernel<<<blocks, TRACE_RAY_BG_CUDA_THREADS>>>(
                grid,
                grad_out.data_ptr<float>(),
                color_cache.data_ptr<float>(),
                rays,
                opt,
                log_transmit.data_ptr<float>(),
                accum.data_ptr<float>(),
                false,
                // Output
                grads);
    }

    CUDA_CHECK_ERRORS;
}

void volume_render_cuvol_fused(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor rgb_gt,
        float beta_loss,
        float sparsity_loss,
        torch::Tensor rgb_out,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    CHECK_INPUT(rgb_gt);
    CHECK_INPUT(rgb_out);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    bool use_background = grid.background_cubemap.size(0) > 0;
    bool need_log_transmit = use_background || beta_loss > 0.f;
    torch::Tensor log_transmit, accum;
    if (need_log_transmit) {
        log_transmit = _get_empty_1d(rays.origins);
    }
    if (use_background) {
        accum = _get_empty_1d(rays.origins);
    }

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_CUDA_THREADS);
        device::render_ray_kernel<<<blocks, TRACE_RAY_CUDA_THREADS>>>(
                grid, rays, opt,
                // Output
                rgb_out.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
                need_log_transmit ? log_transmit.data_ptr<float>() : nullptr);
    }

    if (use_background) {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, TRACE_RAY_BG_CUDA_THREADS);
        device::render_background_kernel<<<blocks, TRACE_RAY_BG_CUDA_THREADS>>>(
                grid,
                rays,
                opt,
                log_transmit.data_ptr<float>(),
                rgb_out.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    }

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_BKWD_CUDA_THREADS);
        device::render_ray_backward_kernel<<<blocks, TRACE_RAY_BKWD_CUDA_THREADS>>>(
                grid,
                rgb_gt.data_ptr<float>(),
                rgb_out.data_ptr<float>(),
                rays, opt,
                true,
                beta_loss > 0.f ? log_transmit.data_ptr<float>() : nullptr,
                beta_loss / Q,
                sparsity_loss,
                // Output
                grads,
                use_background ? accum.data_ptr<float>() : nullptr,
                nullptr);
    }

    if (use_background) {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, TRACE_RAY_BG_CUDA_THREADS);
        device::render_background_backward_kernel<<<blocks, TRACE_RAY_BG_CUDA_THREADS>>>(
                grid,
                rgb_gt.data_ptr<float>(),
                rgb_out.data_ptr<float>(),
                rays,
                opt,
                log_transmit.data_ptr<float>(),
                accum.data_ptr<float>(),
                true,
                // Output
                grads);
    }

    CUDA_CHECK_ERRORS;
}
