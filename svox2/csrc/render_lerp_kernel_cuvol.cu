#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
#include <torch/extension.h>
#include "cuda_util.cuh"
#include "random_util.cuh"
#include "data_spec_packed.cuh"
#include "render_util.cuh"
#include "cubemap_util.cuh"

#include <iostream>
#include <cstdint>
#include <tuple>

namespace {
const int WARP_SIZE = 32;

const int TRACE_RAY_CUDA_THREADS = 128;
const int TRACE_RAY_CUDA_RAYS_PER_BLOCK = TRACE_RAY_CUDA_THREADS / WARP_SIZE;

const int TRACE_RAY_BKWD_CUDA_THREADS = 128;
const int TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK = TRACE_RAY_BKWD_CUDA_THREADS / WARP_SIZE;

const int MIN_BLOCKS_PER_SM = 8;
typedef hipcub::WarpReduce<float> WarpReducef;

namespace device {


// * For ray rendering
__device__ __inline__ void trace_ray_cuvol(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        float* __restrict__ sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        float* __restrict__ out) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;

    if (ray.tmin > ray.tmax && grid.background_nlayers == 0) {
        out[lane_colorgrp] = opt.background_brightness;
        return;
    }

    float t = ray.tmin;
    float outv = 0.f;

    float light_intensity = 0.f;

    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }

        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);

        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }
        float sigma = trilerp_cuvol_one(
                grid.links, grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);
        if (opt.last_sample_opaque && t + opt.step_size > ray.tmax) {
            sigma += 1e9;
        }
        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            lane_color *= sphfunc_val[lane_colorgrp_id];

            const float pcnt = ray.world_step * sigma;
            const float weight = _EXP(light_intensity) * (1.f - _EXP(-pcnt));
            light_intensity -= pcnt;

            float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           lane_color, lane_colorgrp_id == 0);
            outv += weight * fmaxf(lane_color_total + 0.5f, 0.f);  // Clamp to [+0, 1]
            if (_EXP(light_intensity) < opt.stop_thresh) {
                const float renorm_val = 1.f / (1.f - _EXP(light_intensity));
                if (lane_colorgrp_id == 0) {
                    out[lane_colorgrp] *= renorm_val;
                }
                light_intensity = -1e3f;
                break;
            }
        }
        t += opt.step_size;
    }

    if (grid.background_nlayers > 0 && light_intensity > -20.f) {
        // TODO WIP

        ConcentricSpheresIntersector csi(
                grid.size,
                ray.origin,
                ray.dir,
                ray.world_step / opt.step_size);

        // float t_last;
        // const float r_min = fmaxf(_dist_ray_to_origin(csi.origin, csi.dir), opt.background_msi_scale);
        // csi.intersect(r_min + 1e-4f, &t_last);
        // if (csi.dir[0] < 0.0) {
        //     printf("ray_ori=[%f, %f, %f] ray_dir=[%f, %f, %f] ray_w=[%f]\n",
        //             ray.origin[0], ray.origin[1], ray.origin[2],
        //             ray.dir[0], ray.dir[1], ray.dir[2],
        //             ray.world_step
        //           );
        //     printf("csi_ori=[%f, %f, %f] csi_dir=[%f, %f, %f] csi_wss=%f csi_q2a=%f qb=%f f=%f r_min=%f t_last=%f\n",
        //             csi.origin[0], csi.origin[1], csi.origin[2],
        //             csi.dir[0], csi.dir[1], csi.dir[2],
        //             csi.world_step_scale,
        //             csi.q2a,
        //             csi.qb,
        //             csi.f,
        //             r_min,
        //             t_last
        //           );
        // }

        const float* cubemap_data = grid.background_cubemap;
        const int cubemap_step = 6 * grid.background_reso * grid.background_reso * /*n_channels*/ 4;
        for (int i = 0; i < grid.background_nlayers; ++i) {
            const float radius = opt.background_msi_scale * float(grid.background_nlayers) /
                                (float(grid.background_nlayers - i - 0.5f));
            const float thickness = radius - opt.background_msi_scale * float(grid.background_nlayers) /
                                (float(grid.background_nlayers - i));
            float t_inter;
            // if (csi.dir[0] < 0.0) {
            //     printf("i=%d r=%f\n", i, radius);
            // }
            if (csi.intersect(radius, &t_inter)) {
#pragma unroll 3
                for (int j = 0; j < 3; ++j) {
                    ray.pos[j] = fmaf(t_inter, csi.dir[j], csi.origin[j]);
                }
                // if (csi.dir[0] < 0.0) {
                //     printf(" I! t_inter=%f pos=[%f, %f, %f]\n",
                //             t_inter,
                //             ray.pos[0], ray.pos[1], ray.pos[2]);
                // }

                const CubemapCoord coord = dir_to_cubemap_coord(ray.pos,
                                                                grid.background_reso, /* EAC */ true);
                const CubemapBilerpQuery query = cubemap_build_query(coord,
                                                                     grid.background_reso);


                const float sigma = cubemap_sample(cubemap_data,
                                           query,
                                           grid.background_reso,
                                           /*n_channels*/ 4,
                                           3);
                if (sigma > opt.sigma_thresh) {
                    const float group_color = cubemap_sample(cubemap_data,
                            query,
                            grid.background_reso,
                            /*n_channels*/ 4,
                            lane_colorgrp);

                    const float pcnt = csi.world_step_scale * thickness * sigma;
                    const float weight = _EXP(light_intensity) * (1.f - _EXP(-pcnt));
                    light_intensity -= pcnt;
                    // if (csi.dir[0] < 0.0) {
                    //     printf(" wsc=%f, t_inter=%f, t_last=%f, sigma=%f, weight=%f, li=%f\n",
                    //             csi.world_step_scale, t_inter, t_last, sigma, weight, light_intensity);
                    // }

                    outv += weight * fmaxf(group_color + 0.5f, 0.f);  // Clamp to [+0, infty)
                }
                // t_last = t_inter;
            }
            if (cubemap_data != nullptr)
                cubemap_data += cubemap_step;
        }
    }

    outv += _EXP(light_intensity) * opt.background_brightness;
    if (lane_colorgrp_id == 0) {
        out[lane_colorgrp] = outv;
    }
}

__device__ __inline__ void trace_ray_cuvol_backward(
        const PackedSparseGridSpec& __restrict__ grid,
        const float* __restrict__ grad_output,
        const float* __restrict__ color_cache,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        const float* __restrict__ sphfunc_val,
        float* __restrict__ grad_sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        PackedGridOutputGrads& __restrict__ grads
        ) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;
    const uint32_t leader_mask = 1U | (1U << grid.basis_dim) | (1U << (2 * grid.basis_dim));

    if (ray.tmin > ray.tmax && grid.background_nlayers == 0) return;
    float t = ray.tmin;

    const float gout = grad_output[lane_colorgrp];

    float accum = fmaf(color_cache[0], grad_output[0],
                      fmaf(color_cache[1], grad_output[1],
                           color_cache[2] * grad_output[2]));

    float light_intensity = 0.f;

    // remat samples
    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }
        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);
        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }

        float sigma = trilerp_cuvol_one(
                grid.links,
                grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);
        if (opt.last_sample_opaque && t + opt.step_size > ray.tmax) {
            sigma += 1e9;
        }
        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            float weighted_lane_color = lane_color * sphfunc_val[lane_colorgrp_id];

            const float pcnt = ray.world_step * sigma;
            const float weight = _EXP(light_intensity) * (1.f - _EXP(-pcnt));
            light_intensity -= pcnt;

            const float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           weighted_lane_color, lane_colorgrp_id == 0) + 0.5f;
            float total_color = fmaxf(lane_color_total, 0.f);
            float color_in_01 = total_color == lane_color_total;
            total_color *= gout; // Clamp to [+0, 1]

            float total_color_c1 = __shfl_sync(leader_mask, total_color, grid.basis_dim);
            total_color += __shfl_sync(leader_mask, total_color, 2 * grid.basis_dim);
            total_color += total_color_c1;

            color_in_01 = __shfl_sync((1U << grid.sh_data_dim) - 1, color_in_01, lane_colorgrp * grid.basis_dim);
            const float grad_common = weight * color_in_01 * gout;
            const float curr_grad_color = sphfunc_val[lane_colorgrp_id] * grad_common;

            if (grid.basis_type != BASIS_TYPE_SH) {
                float curr_grad_sphfunc = lane_color * grad_common;
                const float curr_grad_up2 = __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, 2 * grid.basis_dim);
                curr_grad_sphfunc += __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, grid.basis_dim);
                curr_grad_sphfunc += curr_grad_up2;
                if (lane_id < grid.basis_dim) {
                    grad_sphfunc_val[lane_id] += curr_grad_sphfunc;
                }
            }

            accum -= weight * total_color;
            float curr_grad_sigma = ray.world_step * (
                    total_color * _EXP(light_intensity) - accum);
            trilerp_backward_cuvol_one(grid.links, grads.grad_sh_out,
                    grid.stride_x,
                    grid.size[2],
                    grid.sh_data_dim,
                    ray.l, ray.pos,
                    curr_grad_color, lane_id);
            if (lane_id == 0) {
                trilerp_backward_cuvol_one_density(
                        grid.links,
                        grads.grad_density_out,
                        grads.mask_out,
                        grid.stride_x,
                        grid.size[2],
                        ray.l, ray.pos, curr_grad_sigma);
            }
            if (_EXP(light_intensity) < opt.stop_thresh) {
                break;
            }
        }
        t += opt.step_size;
    }

    if (grid.background_nlayers > 0 && light_intensity > -20.f &&
        grads.grad_background_out != nullptr) {
        // Performance SUCKS

        ConcentricSpheresIntersector csi(
                grid.size,
                ray.origin,
                ray.dir,
                ray.world_step / opt.step_size);

        // const float r_min = fmaxf(_dist_ray_to_origin(csi.origin, csi.dir), opt.background_msi_scale);
        // float t_last;
        // csi.intersect(r_min + 1e-4f, &t_last);

        const float* cubemap_data = grid.background_cubemap;
        float* grad_cubemap_data = grads.grad_background_out;
        bool* mask_cubemap_ptr = grads.mask_background_out;
        const int cubemap_step = 6 * grid.background_reso * grid.background_reso;
        for (int i = 0; i < grid.background_nlayers; ++i) {
            const float radius = opt.background_msi_scale * float(
                    grid.background_nlayers) / (float(grid.background_nlayers - i - 0.5f));
            const float thickness = radius - opt.background_msi_scale * float(grid.background_nlayers) /
                                (float(grid.background_nlayers - i));
            float t_inter;
            if (csi.intersect(radius, &t_inter)) {
#pragma unroll 3
                for (int j = 0; j < 3; ++j) {
                    ray.pos[j] = fmaf(t_inter, csi.dir[j], csi.origin[j]);
                }

                const CubemapCoord coord = dir_to_cubemap_coord(ray.pos,
                                                                grid.background_reso, /* EAC */ true);
                const CubemapBilerpQuery query = cubemap_build_query(coord,
                                                                     grid.background_reso);


                const float sigma = cubemap_sample(cubemap_data,
                                           query,
                                           grid.background_reso,
                                           /*n_channels*/ 4,
                                           3);
                if (sigma > opt.sigma_thresh) {
                    const float group_color = cubemap_sample(cubemap_data,
                            query,
                            grid.background_reso,
                            /*n_channels*/ 4,
                            lane_colorgrp) + 0.5f;

                    const float pcnt = csi.world_step_scale * thickness * sigma;
                    const float weight = _EXP(light_intensity) * (1.f - _EXP(-pcnt));
                    light_intensity -= pcnt;

                    float total_color = fmaxf(group_color, 0.f);
                    float color_in_01 = total_color == group_color;
                    total_color *= gout;

                    float total_color_c1 = __shfl_sync(leader_mask, total_color, grid.basis_dim);
                    total_color += __shfl_sync(leader_mask, total_color, 2 * grid.basis_dim);
                    total_color += total_color_c1;

                    const float curr_grad_color = weight * color_in_01 * gout;

                    accum -= weight * total_color;
                    float curr_grad_sigma = csi.world_step_scale * thickness * (
                            total_color * _EXP(light_intensity) - accum);

                    if (lane_colorgrp_id == 0) {
                        cubemap_sample_backward(
                                grad_cubemap_data,
                                query,
                                grid.background_reso,
                                4,
                                curr_grad_color,
                                lane_colorgrp);

                        if (lane_id == 0) {
                            cubemap_sample_backward(
                                    grad_cubemap_data,
                                    query,
                                    grid.background_reso,
                                    4,
                                    curr_grad_sigma,
                                    3,
                                    mask_cubemap_ptr);
                        }
                    }
                }

                // t_last = t_inter;
            }
            if (cubemap_data != nullptr)
                cubemap_data += cubemap_step * 4 /* n_channels */;
            if (grad_cubemap_data != nullptr)
                grad_cubemap_data += cubemap_step * 4/* n_channels */;
            if (mask_cubemap_ptr != nullptr)
                mask_cubemap_ptr += cubemap_step;
        }
    }
}


// BEGIN KERNELS

__launch_bounds__(TRACE_RAY_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_kernel(
        PackedSparseGridSpec grid,
        PackedRaysSpec rays,
        RenderOptions opt,
        torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> out) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
            rays.dirs[ray_id].data());
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 ray_spec[ray_blk_id].dir,
                 sphfunc_val[ray_blk_id]);
    if (lane_id == 0) {
        ray_find_bounds(ray_spec[ray_blk_id], grid, opt);
    }
    __syncwarp((1U << grid.sh_data_dim) - 1);

    trace_ray_cuvol(
        grid,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        out[ray_id].data());
}

__launch_bounds__(TRACE_RAY_BKWD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_backward_kernel(
    PackedSparseGridSpec grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
    PackedRaysSpec rays,
    RenderOptions opt,
    PackedGridOutputGrads grads,
    bool grad_out_is_rgb = false) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK][10];
    __shared__ float grad_sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
                             rays.dirs[ray_id].data());
    const float vdir[3] = {ray_spec[ray_blk_id].dir[0],
                     ray_spec[ray_blk_id].dir[1],
                     ray_spec[ray_blk_id].dir[2] };
    if (lane_id < grid.basis_dim) {
        grad_sphfunc_val[ray_blk_id][lane_id] = 0.f;
    }
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 vdir, sphfunc_val[ray_blk_id]);
    if (lane_id == 0) {
        ray_find_bounds(ray_spec[ray_blk_id], grid, opt);
    }

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[ray_id * 3 + i] - grad_output[ray_id * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[ray_id * 3 + i];
        }
    }

    __syncwarp((1U << grid.sh_data_dim) - 1);
    trace_ray_cuvol_backward(
        grid,
        grad_out,
        color_cache + ray_id * 3,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        grad_sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        grads);
    calc_sphfunc_backward(
                 grid, lane_id,
                 ray_id,
                 vdir,
                 sphfunc_val[ray_blk_id],
                 grad_sphfunc_val[ray_blk_id],
                 grads.grad_basis_out);
}

}  // namespace device
}  // namespace

torch::Tensor volume_render_cuvol(SparseGridSpec& grid, RaysSpec& rays, RenderOptions& opt) {
    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();

    const auto Q = rays.origins.size(0);

    const int cuda_n_threads = TRACE_RAY_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads);
    torch::Tensor results = torch::empty_like(rays.origins);
    // printf("CB %d %d\n", cuda_n_threads, blocks);
    device::render_ray_kernel<<<blocks, cuda_n_threads>>>(
            grid, rays, opt,
            // Output
            results.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
    return results;
}

void volume_render_cuvol_backward(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor grad_out,
        torch::Tensor color_cache,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    const int cuda_n_threads_render_backward = TRACE_RAY_BKWD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads_render_backward);

    device::render_ray_backward_kernel<<<blocks,
           cuda_n_threads_render_backward>>>(
            grid,
            grad_out.data_ptr<float>(),
            color_cache.data_ptr<float>(),
            rays, opt,
            // Output
            grads);

    CUDA_CHECK_ERRORS;
}

void volume_render_cuvol_fused(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor rgb_gt,
        torch::Tensor rgb_out,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    CHECK_INPUT(rgb_gt);
    CHECK_INPUT(rgb_out);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_CUDA_THREADS);
        device::render_ray_kernel<<<blocks, TRACE_RAY_CUDA_THREADS>>>(
                grid, rays, opt,
                // Output
                rgb_out.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    }

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_BKWD_CUDA_THREADS);
        device::render_ray_backward_kernel<<<blocks, TRACE_RAY_BKWD_CUDA_THREADS>>>(
                grid,
                rgb_gt.data_ptr<float>(),
                rgb_out.data_ptr<float>(),
                rays, opt,
                // Output
                grads,
                true);
    }

    CUDA_CHECK_ERRORS;
}
